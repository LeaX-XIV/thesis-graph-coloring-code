#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <thrust/count.h>

#include "cudaKernels.h"

#include <algorithm>
#include <iostream>
#include <vector>

#define CUDA_MAX_BLOCKS 2147483647 // Maximum blocks to launch, depending on GPU

__global__ void color_jpl_kernel(int n, int c, const size_t* Ao, const size_t* Ac, const int* randoms, int* colors);

int color_jpl(int const n, const size_t* Ao, const size_t* Ac, int* colors, const int* randoms) {
	hipError_t err = hipSuccess;

	size_t* dAo;
	size_t* dAc;
	int* dRandoms;
	int* dColors;
	
	err = hipMalloc(&dAo, (n+1) * sizeof(*dAo));
	if (err != hipSuccess) {
		std::cout << "Error1: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dAc, Ao[n] * sizeof(*dAc));
	if (err != hipSuccess) {
		std::cout << "Error2: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dRandoms, n * sizeof(*dRandoms));
	if (err != hipSuccess) {
		std::cout << "Error3: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dColors, n * sizeof(*dColors));
	if (err != hipSuccess) {
		std::cout << "Error4: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

	err = hipMemcpy(dAo, Ao, (n+1) * sizeof(*Ao), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error5: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dAc, Ac, Ao[n] * sizeof(*Ac), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error6: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dRandoms, randoms, n * sizeof(*randoms), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error7: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dColors, colors, n * sizeof(*colors), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error8: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	
	int c;
	for (c = 0; c < n; ++c) {
		int nt = 256;
		int nb = std::min((n + nt - 1) / nt, CUDA_MAX_BLOCKS);
		color_jpl_kernel<<<nb, nt>>>(n, c, dAo, dAc, dRandoms, dColors);
		err = hipMemcpy(colors, dColors, n * sizeof(*colors), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			std::cout << "Error9: " << hipGetErrorString(err) << std::endl;
			goto Error;
		}
		int left = (int)thrust::count(colors, colors + n, -1);
		//std::cout << left << std::endl;
		if (left == 0) break;
	}

Error:
	hipFree(dAo);
	hipFree(dAc);
	hipFree(dRandoms);
	hipFree(dColors);

	return err == hipSuccess ? c : -1;
}

__global__ void color_jpl_kernel(int n, int c, const size_t* Ao, const size_t* Ac, const int* randoms, int* colors) {
	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		bool f = true; // true if you have max random

		// ignore nodes colored earlier
		if (colors[i] != -1) continue;

		int ir = randoms[i];

		// look at neighbors to check their random number
		for (int k = Ao[i]; k < Ao[i + 1]; k++) {
			// ignore nodes colored earlier (and yourself)
			int j = Ac[k];
			int jc = colors[j];
			if (((jc != -1) && (jc != c)) || (i == j)) continue;
			int jr = randoms[j];
			if (ir <= jr) f = false;
		}

		// assign color if you have the maximum random number
		if (f) colors[i] = c;
	}
}