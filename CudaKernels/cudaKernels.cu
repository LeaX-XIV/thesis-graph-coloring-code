#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <thrust/count.h>

#include "cudaKernels.h"
#include "hipsparse.h"

#include <algorithm>
#include <iostream>
#include <vector>

#define CUDA_MAX_BLOCKS 2147483647 // Maximum blocks to launch, depending on GPU

__global__ void create_independent_set_kernel(int n, const int* Ao, const int* Ac, const int* randoms, const int* colors, unsigned int* set);
__global__ void expand_to_maximal_independent_set_kernel(int n, const int* Ao, const int* Ac, const int* colors, unsigned int* set);
__global__ void color_jpl_kernel(int n, int c, int* colors, const unsigned int* set);

int color_jpl(int const n, const int* Ao, const int* Ac, int* colors, const int* randoms) {
	hipError_t err = hipSuccess;

	int* dAo;
	int* dAc;
	int* dRandoms;
	int* dColors;
	unsigned int* dSet;

#ifdef COMPUTE_ELAPSED_TIME
	Benchmark& bm = *Benchmark::getInstance();
#endif

	err = hipMalloc(&dAo, (n+1) * sizeof(*dAo));
	if (err != hipSuccess) {
		std::cout << "Error1: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dAc, Ao[n] * sizeof(*dAc));
	if (err != hipSuccess) {
		std::cout << "Error2: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dRandoms, n * sizeof(*dRandoms));
	if (err != hipSuccess) {
		std::cout << "Error3: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dColors, n * sizeof(*dColors));
	if (err != hipSuccess) {
		std::cout << "Error4: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMalloc(&dSet, n * sizeof(*dSet));
	if (err != hipSuccess) {
		std::cout << "Error4: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

	err = hipMemcpy(dAo, Ao, (n+1) * sizeof(*Ao), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error5: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dAc, Ac, Ao[n] * sizeof(*Ac), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error6: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dRandoms, randoms, n * sizeof(*randoms), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error7: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dColors, colors, n * sizeof(*colors), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error8: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemset(dSet, 0x0, n * sizeof(*dSet));
	if (err != hipSuccess) {
		std::cout << "Error8: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

#ifdef COMPUTE_ELAPSED_TIME
	bm.sampleTime();
#endif

	int c;
	int left;
	for (c = 0, left = n; left > 0 && c < n; ++c) {
		int nt = 256;
		int nb = std::min((n + nt - 1) / nt, CUDA_MAX_BLOCKS);
		create_independent_set_kernel<<<nb, nt>>>(n, dAo, dAc, dRandoms, dColors, dSet);
		expand_to_maximal_independent_set_kernel<<<nb, nt>>>(n, dAo, dAc, dColors, dSet);
		color_jpl_kernel<<<nb, nt>>>(n, c, dColors, dSet);
		err = hipMemcpy(colors, dColors, n * sizeof(*colors), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			std::cout << "Error9: " << hipGetErrorString(err) << std::endl;
			goto Error;
		}
		left = (int)thrust::count(colors, colors + n, -1);
	}

Error:
#ifdef COMPUTE_ELAPSED_TIME
	bm.sampleTimeToFlag(1);
#endif
	hipFree(dAo);
	hipFree(dAc);
	hipFree(dRandoms);
	hipFree(dColors);

	return err == hipSuccess ? c : -1;
}

__global__ void create_independent_set_kernel(int n, const int* Ao, const int* Ac, const int* randoms, const int* colors, unsigned int* set) {
	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		//bool f = true; // true if you have max random

		// ignore nodes colored earlier
		if (colors[i] != -1) continue;
		set[i] = 0x1;
		int ir = randoms[i];

		// look at neighbors to check their random number
		for (int k = Ao[i]; k < Ao[i + 1]; k++) {
			// ignore nodes colored earlier (and yourself)
			int j = Ac[k];
			int jc = colors[j];
			if ((jc != -1) || (i == j)) continue;
			int jr = randoms[j];
			if (ir <= jr) set[i] = 0x0;
		}
		// assign color if you have the maximum random number
		//if (f) colors[i] = c;
	}
}

__global__ void expand_to_maximal_independent_set_kernel(int n, const int* Ao, const int* Ac, const int* colors, unsigned int* set) {
	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		// Ignore nodes colored earlier or already in set
		if (colors[i] != -1 || set[i] != 0x0) continue;

		set[i] = 0x2;

		for (int k = Ao[i]; k < Ao[i + 1]; k++) {
			// ignore nodes colored earlier (and yourself)
			int j = Ac[k];
			int jc = colors[j];
			if ((jc != -1) || (i == j)) continue;
			// cannot be part of MIS if neighbor is in initial set
			//  or if neighboring vertex with higher degree is trying to enter the MIS
			if (set[j] == 0x1 ||
				(set[j] == 0x2 && Ao[i+1] - Ao[i] <= Ao[j+1] - Ao[j])
			)
				set[i] = 0x0;
		}

		if (set[i] != 0x0) set[i] = 0x1;
	}
}

__global__ void color_jpl_kernel(int n, int c, int* colors, const unsigned int* set) {
	for (int i = threadIdx.x + blockIdx.x * blockDim.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		if (colors[i] != -1) continue;

		if(set[i] != 0x0) colors[i] = c;
	}
}

int color_cusparse(int const n, const int* Ao, const int* Ac, int* colors) {
	hipError_t err = hipSuccess;

	float* dAv;
	int* dAo;
	int* dAc;
	int* dColors;

#ifdef COMPUTE_ELAPSED_TIME
	Benchmark& bm = *Benchmark::getInstance();
	bm.sampleTime();
#endif

	err = hipMalloc(&dAo, (n + 1) * sizeof(*dAo));
	if (err != hipSuccess) {
		std::cout << "Error1: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dAo, Ao, (n + 1) * sizeof(*Ao), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error5: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

	err = hipMalloc(&dAc, Ao[n] * sizeof(*dAc));
	if (err != hipSuccess) {
		std::cout << "Error2: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dAc, Ac, Ao[n] * sizeof(*Ac), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error6: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

	err = hipMalloc(&dColors, n * sizeof(*dColors));
	if (err != hipSuccess) {
		std::cout << "Error4: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}
	err = hipMemcpy(dColors, colors, n * sizeof(*colors), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cout << "Error8: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

	err = hipMalloc(&dAv, Ao[n] * sizeof(*dAv));
	if (err != hipSuccess) {
		std::cout << "Error1: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}	

	int c;
	float fractionToColor = 1.0;

	hipsparseStatus_t status;
	hipsparseHandle_t handle;
	hipsparseMatDescr_t matrixDesc;
	hipsparseColorInfo_t colorInfo;

	status = hipsparseCreate(&handle);
	status = hipsparseCreateMatDescr(&matrixDesc);
	status = hipsparseCreateColorInfo(&colorInfo);


#ifdef COMPUTE_ELAPSED_TIME
	bm.sampleTimeToFlag(1);
#endif
	status = hipsparseScsrcolor(handle,
		n,
		Ao[n],
		matrixDesc,
		dAv,
		dAo,
		dAc,
		&fractionToColor,
		&c,
		dColors,
		NULL,
		colorInfo);

#ifdef COMPUTE_ELAPSED_TIME
	hipDeviceSynchronize();
	bm.sampleTimeToFlag(2);
#endif

	err = hipMemcpy(colors, dColors, n * sizeof(*colors), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		std::cout << "Error9: " << hipGetErrorString(err) << std::endl;
		goto Error;
	}

Error:
#ifdef COMPUTE_ELAPSED_TIME
	bm.sampleTimeToFlag(3);
#endif
	hipFree(dAv);
	hipFree(dAo);
	hipFree(dAc);
	hipFree(dColors);

	hipsparseDestroyMatDescr(matrixDesc);
	hipsparseDestroyColorInfo(colorInfo);
	hipsparseDestroy(handle);

	return err == hipSuccess ? c : -1;
}